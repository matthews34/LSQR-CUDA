#include "hip/hip_runtime.h"
#include "GPUVector.h"
#include "utils.h"


__global__ void norm_kernel(const double *data, const int n, double *output) {

	__shared__ double partialSum[2*BLOCK_SIZE];

	int i = threadIdx.x;

	int start = 2 * blockIdx.x * blockDim.x;

	// Each thread loads two elements
	partialSum[i] = data[start + i] * data[start + i];
	partialSum[blockDim.x + i] = data[start + blockDim.x + i] * data[start + blockDim.x + i];

	for (int stride = blockDim.x; stride > 0; stride /= 2) {
		__syncthreads();
		if (i < stride) partialSum[i] += partialSum[i + stride];
	}

	if (i == 0) *output = sqrt(partialSum[0]);
}

double GPUVector::norm() {
	double h_result;
	double *d_result;

	dim3 dimBlock(BLOCK_SIZE);
	dim3 dimGrid((n + dimBlock.x - 1) / dimBlock.x);

	hipMalloc(&d_result, sizeof(double));

	norm_kernel<<<dimGrid, dimBlock>>>(elements, n, d_result);

	hipMemcpy(&h_result, d_result, sizeof(double), hipMemcpyDeviceToHost);
	CUDAFREE(d_result);

	return h_result;
}

__global__ void scale_kernel(const double *input, double *output, const int n, const double s) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	output[i] = input[i] * s;
}

GPUVector operator*(const GPUVector v, const double s) {
	dim3 dimBlock(BLOCK_SIZE);
	dim3 dimGrid((v.n + dimBlock.x - 1) / dimBlock.x);

	GPUVector c(v.n);
	
	scale_kernel<<<dimGrid, dimBlock>>>(v.elements, c.elements, v.n, s);

	return c;
}
GPUVector operator*(const double s, const GPUVector v) {
	return v * s;
}

__global__ void add_kernel(const double *a, const double *b, double *c) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	c[i] = a[i] + b[i];
}

__global__ void sub_kernel(const double *a, const double *b, double *c) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	c[i] = a[i] - b[i];
}

GPUVector GPUVector::operator+(const GPUVector b) {
	dim3 dimBlock(BLOCK_SIZE);
	dim3 dimGrid((n + dimBlock.x - 1) / dimBlock.x);

	GPUVector c(n);
	
	add_kernel<<<dimGrid, dimBlock>>>(elements, b.elements, c.elements);

	return c;
}

GPUVector GPUVector::operator-(const GPUVector b) {
	dim3 dimBlock(BLOCK_SIZE);
	dim3 dimGrid((n + dimBlock.x - 1) / dimBlock.x);

	GPUVector c(n);

	sub_kernel<<<dimGrid, dimBlock>>>(elements, b.elements, c.elements);

	return c;
}