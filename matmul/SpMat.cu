#include "hip/hip_runtime.h"
#include "SpMat.h"
#include "utils.h"
#include "stdio.h"

__global__ void nnz_in_row(const double* data_partial, const int n, const int cols, int* nnz) {
	int i = threadIdx.x + threadIdx.y * blockDim.x + blockIdx.x * blockDim.x * blockDim.y;
	if(n < i)
		return;
	if(abs(data_partial[i]) > ZERO)
		atomicAdd(nnz, 1);
}

__global__ void cum_sum(const int * rowNnz, const int rows, int * cumsum) {
	int idx = threadIdx.x + threadIdx.y * blockDim.x + blockIdx.x * blockDim.x * blockDim.y;
	if(rows < idx)
		return;
	if(idx == rows-1)
		cumsum[0] = 0;
	else
		cumsum[idx+1] = rowNnz[idx];
	for(int stride = 1; stride < rows; stride*=2) {
		__syncthreads();
		if(stride < idx)
			cumsum[idx] = cumsum[idx] + cumsum[idx-stride];
	}
	__syncthreads();
	if(idx == rows)
		cumsum[idx] = cumsum[idx] + rowNnz[idx-1];
}

__global__ void get_ind_val(const double* data_partial, const int n, const int cols, int * colInd, double * val, int& nnz) {
	int i = threadIdx.x + threadIdx.y * blockDim.x + blockIdx.x * blockDim.x * blockDim.y;
	//int elem_num = nnz;
	if(n < i)
		return;
	if(!(abs(data_partial[i]) > ZERO))
		return;
	int my_ind;
	my_ind = atomicSub(&nnz, 1) - 1;
	colInd[my_ind] = i;
	val[my_ind] = data_partial[i];
}

SpMat::SpMat(int rows, int cols, double * data) : rows(rows), cols(cols) {
	//dim3 dimGrid((cols + dimBlock.x - 1) / dimBlock.x);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(dimBlock.x*dimBlock.y / (cols*BULK_SIZE) + 1);
	double * data_partial;
	int *RowNonzero;
	int *nnz_elem;
	hipMalloc(&nnz_elem, sizeof(int));
	hipMalloc(&data_partial, cols * sizeof(double));
	hipMalloc(&RowNonzero, rows*sizeof(int));
	for(int i = 0; i < rows; i++) {
		hipMemset(nnz_elem,0,sizeof(int));
		hipMemcpy(data_partial, data + cols * i, cols * sizeof(double), hipMemcpyDefault);
		nnz_in_row<<<dimGrid, dimBlock>>>(data_partial, cols, cols, nnz_elem);
		hipMemcpy((RowNonzero + i), nnz_elem, sizeof(int),hipMemcpyDefault);
	}
	hipMalloc(&rowPtr, (rows + 1) * sizeof(int));
	cum_sum<<<dimGrid, dimBlock>>>(RowNonzero, rows, rowPtr);
	hipMemcpy(&nnz,rowPtr + rows,sizeof(int),hipMemcpyDefault);
	hipMalloc(&colInd, (nnz) * sizeof(int));
	hipMalloc(&val, (nnz) * sizeof(double));
	printf("Matrix has %d Non-Zero Elements\n",nnz);
	for(int i = 0; i < rows; i++) {	
		int offset;
		hipMemcpy(&offset, rowPtr + i, sizeof(int), hipMemcpyDefault);
		hipMemcpy(data_partial, data + cols*i, cols * sizeof(double), hipMemcpyDefault);
		get_ind_val<<<dimGrid, dimBlock>>>(data_partial, cols, cols, colInd + offset, val + offset, RowNonzero[i]);
	}
	CUDAFREE(RowNonzero);
	CUDAFREE(data_partial);
}

__global__ void dot_kernel(	const int * rowPtr, const int * colInd, const double* val, 
							const double* x, double* y, int row_num, int col_num, double * y_nnz){
	int idx = threadIdx.x + threadIdx.y * blockDim.x + blockIdx.x * blockDim.x * blockDim.y;
	if(rowPtr[row_num] - 1 < idx)
		return;
	int row;
	for (int i = 0; i < row_num; i++)
		if((idx >= rowPtr[i] && idx < rowPtr[i+1]) && rowPtr[i] != rowPtr[i+1]) {
			row = i;
		}
	y_nnz[idx] = x[colInd[idx]]*val[idx];
	if(idx != rowPtr[row])
		return;
	int n = rowPtr[row+1] - rowPtr[row];
	for(int i = 0; i < n; i++)
		y[row] += y_nnz[idx+i];
}

void SpMat::dot(GPUVector & x,GPUVector & y ) {
	// create nnz threads
	assert(x.n == cols);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(dimBlock.x*dimBlock.y/nnz + 1);
	double *y_nnz;
	hipMalloc(&y_nnz, nnz*sizeof(double));
	dot_kernel<<<dimGrid, dimBlock>>>(rowPtr, colInd, val, x.elements, y.elements, rows, cols, y_nnz);
	CUDAFREE(y_nnz);
}

__global__ void transpose_row_nnz(const int * colInd, int cols, int nnz, int* colNnz, int* cumsum) {
	int idx = threadIdx.x + threadIdx.y * blockDim.x + blockIdx.x * blockDim.x * blockDim.y;
	if(nnz - 1 < idx)
		return;
	atomicAdd(colNnz + colInd[idx],1);
	if(cols < idx)
		return;
	if(idx == cols-1)
		cumsum[0] = 0;
	else
		cumsum[idx+1] = colNnz[idx];
	for(int stride = 1; stride < cols; stride*=2) {
		__syncthreads();
		if(stride < idx)
			cumsum[idx] = cumsum[idx] + cumsum[idx-stride];
	}
	__syncthreads();
	if(idx == cols)
		cumsum[idx] = cumsum[idx] + colNnz[idx-1];
}

__global__ void transpose_kernel(	const int* rowPtr, const int * colInd, const double* val, 
									int * rowInd, double* trans_val,
									int row_num, int col_num, int nnz, int* colNnz, int* colPtr) {
	int idx = threadIdx.x + threadIdx.y * blockDim.x + blockIdx.x * blockDim.x * blockDim.y;
	if(nnz - 1 < idx)
		return;
	int row;
	for (int i = 0; i < row_num; i++)
		if((idx >= rowPtr[i] && idx < rowPtr[i+1]) && rowPtr[i] != rowPtr[i+1]) {
			row = i;
		}
	int my_ind = atomicSub(colNnz + colInd[idx], 1) - 1;
	rowInd[colPtr[colInd[idx]] + my_ind] = row;
	trans_val[colPtr[colInd[idx]] + my_ind] = val[idx];
}

SpMat::SpMat(SpMat &A) : rows(A.cols), cols(A.rows), nnz(A.nnz) {
	hipMalloc(&rowPtr,(rows+1)*sizeof(int));
	hipMalloc(&colInd,nnz*sizeof(int));
	hipMalloc(&val,nnz*sizeof(double));
	int *rowNnz;
	hipMalloc(&rowNnz,rows*sizeof(int));
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(dimBlock.x*dimBlock.y/nnz + 1);
	transpose_row_nnz<<<dimGrid, dimBlock>>>(A.colInd, A.cols, nnz, rowNnz, rowPtr);
	transpose_kernel<<<dimGrid, dimBlock>>>(A.rowPtr, A.colInd, A.val, colInd, val, A.rows, A.cols, A.nnz, rowNnz, rowPtr);
	CUDAFREE(rowNnz);
}

SpMat::~SpMat() {
	CUDAFREE(rowPtr);
	CUDAFREE(colInd);
	CUDAFREE(val);
}