#include "hip/hip_runtime.h"
#include <iostream>
#include "GPUMatrix.h"
#include "GPUVector.h"
#include "CSRMatrix.h"
#include "utils.h"

int main()
{
	// m  is pitch of the matrix resulting in B_t as default
    int m = 6, n = 6;
	// initialize values
	double B_t[] = { 10.0, 0.0, 0.0, 0.0, -2.0, 0.0, 
					3.0, 9.0, 0.0, 0.0, 0.0, 3.0, 
					0.0, 7.0, 8.0, 7.0, 0.0, 0.0, 
					0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 
					0.0, 8.0, 0.0, 9.0, 9.0, 13.0,  
					0.0, 4.0, 0.0, 0.0, 2.0, -1.0};
	double x0[] = {1.0, 2.0, 3.0, 4.0, 5.0, 7.0,9,10}; 
	double *y0 = new double[m];	
	
	//set cusparse handles
	hipsparseHandle_t cusparseHandle;
	hipsparseCreate(&cusparseHandle);
	
	//create GPUMatrix and CSRMatrix from it
	GPUMatrix B_t_gpu(n,m,B_t);
	CSRMatrix B_t_csr(B_t_gpu,cusparseHandle);
	
	//create GPUVectors
	GPUVector x(n,x0);
	GPUVector y(m,y0);
	
	// get transpose matrix of B_t which is B
	CSRMatrix B_csr = B_t_csr.transpose();

	//calculate dot product and write it to y
	std::cout << "Calculating B*x\n";
	B_csr.dot(x, y);

	//create y_cpu for loading the results from gpu
	double *y_cpu = new double[m];
	from_gpu(y_cpu,y.elements,m);
	std::cout << "B*x = (";
	for(int i = 0; i < m-1; i++)
		std::cout << y_cpu[i] << " ";
	std::cout << y_cpu[m-1] << ")" << std::endl;

	//calculate dot product and write it to y
	std::cout << "Calculating B'*x\n";
	B_t_csr.dot(x, y);
	
	from_gpu(y_cpu,y.elements,m);
	std::cout << "B'*x = (";
	for(int i = 0; i < m-1; i++)
		std::cout << y_cpu[i] << " ";
	std::cout << y_cpu[m-1] << ")" << std::endl;
	
	FREE(y_cpu);
	FREE(y0);
	CUSPARSEFREEHANDLE(cusparseHandle);
    hipDeviceReset();
}