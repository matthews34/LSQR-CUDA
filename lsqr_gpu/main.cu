#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <math.h>
#include <chrono>
//#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "matmul/SpMat.h"
#include "matmul/GPUVector.h"
#include "lsqr_gpu.h"
#include <hipblas.h>

double norm(GPUVector &v) {
    double norm = v.norm();
	return norm;
}
// reads vector from file
void read_vector(char* file_name, double** data, int &n) {
	FILE *file = fopen(file_name, "rb");
	if (file==NULL) {fputs ("File error\n",stderr); exit (1);}
	
	char *token = strtok(file_name, "_");
	token = strtok(NULL, "_");
	n = std::stoi( token );
	
	// printf("Vector size: %d\n",n);
	
	*data = (double*) malloc (sizeof(double) * n);
	if (*data == NULL) {fputs ("Memory error",stderr); exit (2);}
	fread(*data,sizeof(double),n,file);
	fclose(file);
}

// reads matrix from file and parses it to csr format
void read_sparse_matrix(char* file_name, int** rowPtr, int** colInd, double** val, int& n, int& m, int& totalNnz) {
	// printf("file_name = %s\n",file_name);
	
	FILE *file = fopen(file_name, "rb");
	if (file==NULL) {fputs ("File error\n",stderr); exit (1);}
	
	char *token = strtok(file_name, "_");
	token = strtok(NULL, "_");
	m = std::stoi( token );
	token = strtok(NULL, "_");
	n = std::stoi( token );
	
	// printf("Matrix size: %dx%d\n",m,n);
	
	double *data = (double*) malloc (sizeof(double) * n);
	int * rowNnz = (int*) malloc(sizeof(int)*m);
	*rowPtr = (int*) malloc(sizeof(int)*(m+1)) ;
	//rowPtr = new int[m];
	// (*rowPtr)[0] = 0;
	totalNnz = 0;
	int rowCounter = 0;
	(*rowPtr)[0] = 0;
	if (data == NULL) {fputs ("Memory error",stderr); exit (2);}
	while(fread(data,sizeof(double),n,file)) {
		rowNnz[rowCounter] = 0;
		for(int i = 0; i < n; i++)
			if(std::abs(data[i]) > ZERO)
				rowNnz[rowCounter]++;
		totalNnz += rowNnz[rowCounter];
		rowCounter++;
		(*rowPtr)[rowCounter] = totalNnz;
	}
	
	// printf("Total Non-Zero Elements: %d\n",totalNnz);	
	rewind(file);

	*val = (double*) malloc(sizeof(double)*totalNnz);
	*colInd = (int*) malloc(sizeof(int)*totalNnz);
	int counter = 0;
	
	while(fread(data,sizeof(double),n,file)) {
		for(int i = 0; i < n; i++) 
			if(std::abs(data[i]) > ZERO){
				(*val)[counter] = data[i];
				(*colInd)[counter] = i;
				counter++;
			}
	}
	fclose(file);
	// printf("Read Data\n");
	FREE(data);
	FREE(rowNnz);
}


// expected input: mxn matrix binary file named "matrix_m_n", m vector binary file named "vector_m"
int main(int argc, char *argv[])
{
	hipblasHandle_t handle;
	hipblasStatus_t status = hipblasCreate(&handle);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		printf("Error creating handle\n");
		exit(-1);
	}
	if(argc < 3) {
		printf("Matrix and vector file required\n");
		return 0;
	}
	char* matrix_file_name = argv[1];
	char* vector_file_name = argv[2];
	int* rowPtr = NULL; 
	int* colInd = NULL; 
	double* val = NULL;
	int n; 
	int m;
	int totalNnz;
	// reads matrix in csr format from file
	read_sparse_matrix(matrix_file_name, &rowPtr, &colInd, &val, n, m, totalNnz);
	double *vec_data = NULL;
	int vec_dim;
	// reads vector from file
	read_vector(vector_file_name, &vec_data, vec_dim);
	if(vec_dim != m) {
		printf("Vector dimension (%d) must agree with number of rows (%d) in matrix",vec_dim,m);
		return 0;
	}
	GPUVector b(handle, vec_dim,vec_data);
	GPUVector x(handle, n);
	SpMat A(rowPtr, colInd, val, m, n, totalNnz);

	printf("Starting Calculation (n = %d,m = %d)\n",n,m);
    printf("initial residual = %f\n",norm(b));
	// Start GPU timing
    hipEvent_t evStart, evStop;
	hipEventCreate(&evStart);
	hipEventCreate(&evStop);
	hipEventRecord(evStart, 0);

	lsqr(A,b,x);

	// Stop GPU timing
	hipEventRecord(evStop, 0);
	hipEventSynchronize(evStop);
	float elapsedTime_ms;
	hipEventElapsedTime(&elapsedTime_ms, evStart, evStop);
	hipEventDestroy(evStart);
	hipEventDestroy(evStop);
	
	double *x_cpu = new double[n];
	hipMemcpy(x_cpu, x.elements, sizeof(double) * n, hipMemcpyDeviceToHost);

	printf("elapsed time [s]: %f\n",elapsedTime_ms/1000);
	GPUVector residual_vec = dot(A,x) - b;
    printf("final residual = %f\n",norm(residual_vec));
	
	printf("x = (");
	for(int i = 0; i < n; i++)
		printf("%f ",x_cpu[i]);
	printf(")\n");

	FREE(x_cpu);	
	FREE(rowPtr);
	FREE(colInd);
	FREE(val);
	FREE(vec_data);
    return 0;
}
