#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <math.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "matmul/SpMat.h"
#include "matmul/GPUVector.h"
#include "lsqr_gpu.h"
#include <hipblas.h>
#include <hipsparse.h>

// method for computing the norm of a vector
double norm(GPUVector &v) {
    double norm = v.norm();
	return norm;
}
// reads vector from file
void read_vector(char* file_name, double** data, int &n) {
	FILE *file = fopen(file_name, "rb");
	if (file==NULL) {fputs ("File error\n",stderr); exit (1);}
	// get array dimensions from the file name
	char *token = strtok(file_name, "_");
	token = strtok(NULL, "_");
	n = std::stoi( token );
	// read all data from file and load it to the vector memory
	*data = (double*) malloc (sizeof(double) * n);
	if (*data == NULL) {fputs ("Memory error",stderr); exit (2);}
	fread(*data,sizeof(double),n,file);
	fclose(file);
}

// reads matrix from file and parses it to csr format
void read_sparse_matrix(char* file_name, int** rowPtr, int** colInd, double** val, int& n, int& m, int& totalNnz) {
	FILE *file = fopen(file_name, "rb");
	if (file==NULL) {fputs ("File error\n",stderr); exit (1);}	
	// read matrix dimensions from file name
	char *token = strtok(file_name, "_");
	token = strtok(NULL, "_");
	m = std::stoi( token );
	token = strtok(NULL, "_");
	n = std::stoi( token );
	// create arrays for 
	double *data = (double*) malloc (sizeof(double) * n);
	int * rowNnz = (int*) malloc(sizeof(int)*m);
	*rowPtr = (int*) malloc(sizeof(int)*(m+1));
	totalNnz = 0;
	int rowCounter = 0;
	(*rowPtr)[0] = 0;
	if (data == NULL) {fputs ("Memory error",stderr); exit (2);}
	// read each line of the file and count the nnz elements per row
	while(fread(data,sizeof(double),n,file)) {
		rowNnz[rowCounter] = 0;
		for(int i = 0; i < n; i++)
			if(std::abs(data[i]) > ZERO)
				rowNnz[rowCounter]++;
		totalNnz += rowNnz[rowCounter];
		rowCounter++;
		(*rowPtr)[rowCounter] = totalNnz;
	}
	// re-read the file and fill values with according column indexes
	rewind(file);
	*val = (double*) malloc(sizeof(double)*totalNnz);
	*colInd = (int*) malloc(sizeof(int)*totalNnz);
	int counter = 0;
	while(fread(data,sizeof(double),n,file)) {
		for(int i = 0; i < n; i++) 
			if(std::abs(data[i]) > ZERO){
				(*val)[counter] = data[i];
				(*colInd)[counter] = i;
				counter++;
			}
	}
	fclose(file);
	FREE(data);
	FREE(rowNnz);
}

// expected input: mxn matrix binary file named "matrix_m_n", m vector binary file named "vector_m"
int main(int argc, char *argv[])
{
	hipblasHandle_t handle;
	hipblasStatus_t status = hipblasCreate(&handle);
    hipsparseHandle_t cusparseH;
	hipsparseStatus_t cusparseStat = hipsparseCreate(&cusparseH);
	assert(cusparseStat == HIPSPARSE_STATUS_SUCCESS);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		printf("Error creating handle\n");
		exit(-1);
	}
	if(argc < 3) {
		printf("Matrix and vector file required\n");
		return 0;
	}
	char* matrix_file_name = argv[1];
	char* vector_file_name = argv[2];
	int* rowPtr = NULL; 
	int* colInd = NULL; 
	double* val = NULL;
	int n; 
	int m;
	int totalNnz;
	// reads matrix in csr format from file
	read_sparse_matrix(matrix_file_name, &rowPtr, &colInd, &val, n, m, totalNnz);
	double *vec_data = NULL;
	int vec_dim;
	// reads vector from file
	read_vector(vector_file_name, &vec_data, vec_dim);
	if(vec_dim != m) {
		printf("Vector dimension (%d) must agree with number of rows (%d) in matrix",vec_dim,m);
		return 0;
	}
	GPUVector b(handle, vec_dim,vec_data);
	GPUVector x(handle, n);
	SpMat A(rowPtr, colInd, val, m, n, totalNnz, cusparseH);

	printf("Starting Calculation (n = %d,m = %d)\n",n,m);
	// Start GPU timing
    hipEvent_t evStart, evStop;
	hipEventCreate(&evStart);
	hipEventCreate(&evStop);
	hipEventRecord(evStart, 0);

	lsqr(A,b,x);
	
	// Stop GPU timing
	hipEventRecord(evStop, 0);
	hipEventSynchronize(evStop);
	float elapsedTime_ms;
	hipEventElapsedTime(&elapsedTime_ms, evStart, evStop);
	hipEventDestroy(evStart);
	hipEventDestroy(evStop);
	
	// get resulting vector
	double *x_cpu = new double[n];
	hipMemcpy(x_cpu, x.elements, sizeof(double) * n, hipMemcpyDeviceToHost);

	printf("elapsed time [s]: %f\n",elapsedTime_ms/1000);
	GPUVector residual_vec = dot(A,x) - b;
    printf("final residual = %f\n",norm(residual_vec));
	
	// print vector
	printf("x = (");
	for(int i = 0; i < n; i++)
		printf("%f ",x_cpu[i]);
	printf(")\n");

	FREE(x_cpu);	
	FREE(rowPtr);
	FREE(colInd);
	FREE(val);
	FREE(vec_data);
    return 0;
}
